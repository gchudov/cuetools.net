
#include <hip/hip_runtime.h>
/**
 * CUETools.FlaCuda: FLAC audio encoder using CUDA
 * Copyright (c) 2009 Gregory S. Chudov
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#ifndef _FLACUDA_KERNEL_H_
#define _FLACUDA_KERNEL_H_

typedef enum
{
    Constant = 0,
    Verbatim = 1,
    Fixed = 8,
    LPC = 32
} SubframeType;

typedef struct
{
    int residualOrder; // <= 32
    int samplesOffs;
    int shift;
    int cbits;
    int size;
    int type;
    int obits;
    int blocksize;
    int best_index;
    int channel;
    int residualOffs;
    int wbits;
    int abits;
    int porder;
    int reserved[2];
} FlaCudaSubframeData;

typedef struct
{
    FlaCudaSubframeData data;
    int coefs[32]; // fixme: should be short?
} FlaCudaSubframeTask;

#define SUM16(buf,tid,op)   buf[tid] op buf[tid + 8]; buf[tid] op buf[tid + 4]; buf[tid] op buf[tid + 2]; buf[tid] op buf[tid + 1];
#define SUM32(buf,tid,op)   buf[tid] op buf[tid + 16]; SUM16(buf,tid,op)
#define SUM64(buf,tid,op)   if (tid < 32) buf[tid] op buf[tid + 32]; __syncthreads(); if (tid < 32) { SUM32(buf,tid,op) }
#define SUM128(buf,tid,op)  if (tid < 64) buf[tid] op buf[tid + 64]; __syncthreads(); SUM64(buf,tid,op)
#define SUM256(buf,tid,op)  if (tid < 128) buf[tid] op buf[tid + 128]; __syncthreads(); SUM128(buf,tid,op)
#define SUM512(buf,tid,op)  if (tid < 256) buf[tid] op buf[tid + 256]; __syncthreads(); SUM256(buf,tid,op)

#define FSQR(s) ((s)*(s))
#define FASTMUL(a,b) __mul24(a,b)

extern "C" __global__ void cudaStereoDecorr(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
	samples[2 * offset + pos] = (s.x + s.y) >> 1;
	samples[3 * offset + pos] = s.x - s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr2(
    int *samples,
    short2 *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
    {
	short2 s = src[pos];
	samples[pos] = s.x;
	samples[1 * offset + pos] = s.y;
    }
}

extern "C" __global__ void cudaChannelDecorr(
    int *samples,
    short *src,
    int offset
)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < offset)
	samples[blockIdx.y * offset + pos] = src[pos * gridDim.y + blockIdx.y];
}

extern "C" __global__ void cudaFindWastedBits(
    FlaCudaSubframeTask *tasks,
    int *samples,
    int tasksPerChannel
)
{
    __shared__ struct {
	volatile int wbits[256];
	volatile int abits[256];
	FlaCudaSubframeData task;
    } shared;

    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.x * tasksPerChannel].data))[threadIdx.x];
    shared.wbits[threadIdx.x] = 0;
    shared.abits[threadIdx.x] = 0;
    __syncthreads();

    for (int pos = 0; pos < shared.task.blocksize; pos += blockDim.x)
    {
	int smp = pos + threadIdx.x < shared.task.blocksize ? samples[shared.task.samplesOffs + pos + threadIdx.x] : 0;
	shared.wbits[threadIdx.x] |= smp;
	shared.abits[threadIdx.x] |= smp ^ (smp >> 31);
    }
    __syncthreads();
    SUM256(shared.wbits, threadIdx.x, |=);
    SUM256(shared.abits, threadIdx.x, |=);
    if (threadIdx.x == 0)
	shared.task.wbits = max(0,__ffs(shared.wbits[0]) - 1);
    if (threadIdx.x == 0)
	shared.task.abits = 32 - __clz(shared.abits[0]) - shared.task.wbits;
    __syncthreads();

    if (threadIdx.x < tasksPerChannel)
	tasks[blockIdx.x * tasksPerChannel + threadIdx.x].data.wbits = shared.task.wbits;
    if (threadIdx.x < tasksPerChannel)
	tasks[blockIdx.x * tasksPerChannel + threadIdx.x].data.abits = shared.task.abits;
}

extern "C" __global__ void cudaComputeAutocor(
    float *output,
    const int *samples,
    const float *window,
    FlaCudaSubframeTask *tasks,
    const int max_order, // should be <= 32
    const int windowCount, // windows (log2: 0,1)
    const int taskCount // tasks per block
)
{
    __shared__ struct {
	float data[512];
	volatile float product[256];
	FlaCudaSubframeData task;
	volatile float result[33];
	volatile int dataPos;
	volatile int dataLen;
	volatile int windowOffs;
	volatile int samplesOffs;
	//volatile int resultOffs;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y * 32);
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + __mul24(taskCount, blockIdx.y >> windowCount)))[tid];
    if (tid == 0) 
    {
	shared.dataPos = __mul24(blockIdx.x, 15 * 32);
	shared.windowOffs = __mul24(blockIdx.y & ((1 << windowCount)-1), shared.task.blocksize) + shared.dataPos;
	shared.samplesOffs = shared.task.samplesOffs + shared.dataPos;
	shared.dataLen = min(shared.task.blocksize - shared.dataPos, 15 * 32 + max_order);
    }
    //if (tid == 32)
	//shared.resultOffs = __mul24(blockIdx.x + __mul24(blockIdx.y, gridDim.x), max_order + 1);
    __syncthreads();

    // fetch samples
    shared.data[tid] = tid < shared.dataLen ? samples[shared.samplesOffs + tid] * window[shared.windowOffs + tid]: 0.0f;
    int tid2 = tid + 256;
    shared.data[tid2] = tid2 < shared.dataLen ? samples[shared.samplesOffs + tid2] * window[shared.windowOffs + tid2]: 0.0f;
    __syncthreads();

    const int ptr = __mul24(threadIdx.x, 15);
    for (int lag = threadIdx.y; lag <= max_order; lag += 8)
    {
        //const int productLen = min(shared.task.blocksize - blockIdx.x * partSize - lag, partSize);
	const int ptr2 = ptr + lag;
	shared.product[tid] =
	    shared.data[ptr + 0] * shared.data[ptr2 + 0] +
	    shared.data[ptr + 1] * shared.data[ptr2 + 1] +
	    shared.data[ptr + 2] * shared.data[ptr2 + 2] +
	    shared.data[ptr + 3] * shared.data[ptr2 + 3] +
	    shared.data[ptr + 4] * shared.data[ptr2 + 4] +
	    shared.data[ptr + 5] * shared.data[ptr2 + 5] +
	    shared.data[ptr + 6] * shared.data[ptr2 + 6] +
	    shared.data[ptr + 7] * shared.data[ptr2 + 7] +
	    shared.data[ptr + 8] * shared.data[ptr2 + 8] +
	    shared.data[ptr + 9] * shared.data[ptr2 + 9] +
	    shared.data[ptr + 10] * shared.data[ptr2 + 10] +
	    shared.data[ptr + 11] * shared.data[ptr2 + 11] +
	    shared.data[ptr + 12] * shared.data[ptr2 + 12] +
	    shared.data[ptr + 13] * shared.data[ptr2 + 13] +
	    shared.data[ptr + 14] * shared.data[ptr2 + 14];
	shared.product[tid] = shared.product[tid] + shared.product[tid + 8] + shared.product[tid + 16] + shared.product[tid + 24];
	shared.product[tid] = shared.product[tid] + shared.product[tid + 2] + shared.product[tid + 4] + shared.product[tid + 6];
	// return results
	if (threadIdx.x == 0)
	    shared.result[lag] = shared.product[tid] + shared.product[tid + 1];
    }
    __syncthreads();
    if (tid <= max_order)
	output[__mul24(blockIdx.x + __mul24(blockIdx.y, gridDim.x), max_order + 1) + tid] = shared.result[tid];
}

extern "C" __global__ void cudaComputeLPC(
    FlaCudaSubframeTask *tasks,
    int taskCount, // tasks per block
    float*autoc,
    int max_order, // should be <= 32
    float *lpcs,
    int windowCount,
    int partCount
)
{
    __shared__ struct {
	FlaCudaSubframeData task;
	volatile float parts[32];
	volatile float ldr[32];
	volatile float gen1[32];
	volatile float error[32];
	volatile float autoc[33];
	volatile int lpcOffs;
	volatile int autocOffs;
    } shared;
    const int tid = threadIdx.x;// + threadIdx.y * 32;
    
    // fetch task data
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(tasks + blockIdx.y * taskCount))[tid];
    if (tid == 0)
    {
	shared.lpcOffs = __mul24(blockIdx.x + __mul24(blockIdx.y, windowCount), max_order + 1) * 32;
	shared.autocOffs = __mul24(__mul24(blockIdx.x + __mul24(blockIdx.y, gridDim.x), max_order + 1), partCount);
    }
    //__syncthreads();
    
    // add up autocorrelation parts

 //   for (int order = threadIdx.x; order <= max_order; order += 32)
 //   {
	//float sum = 0.0f;
	//for (int pos = 0; pos < partCount; pos++)
	//    sum += autoc[shared.autocOffs + pos * (max_order + 1) + order];
	//shared.autoc[order] = sum;
 //   }

    for (int order = 0; order <= max_order; order ++)
    {
	shared.parts[tid] = 0.0f;
	for (int pos = threadIdx.x; pos < partCount; pos += 32)
	    shared.parts[tid] += autoc[shared.autocOffs + pos * (max_order + 1) + order];
	shared.parts[tid] = shared.parts[tid] + shared.parts[tid + 8] + shared.parts[tid + 16] + shared.parts[tid + 24];
	shared.parts[tid] = shared.parts[tid] + shared.parts[tid + 2] + shared.parts[tid + 4] + shared.parts[tid + 6];
	if (threadIdx.x == 0)
	    shared.autoc[order] = shared.parts[tid] + shared.parts[tid + 1];
    }
    //__syncthreads();

    // Compute LPC using Schur and Levinson-Durbin recursion
    if (threadIdx.y == 0)
    {
	float gen0 = shared.gen1[threadIdx.x] = shared.autoc[threadIdx.x+1];
	shared.ldr[threadIdx.x] = 0.0f;
	float error = shared.autoc[0];
	for (int order = 0; order < max_order; order++)
	{
	    // Schur recursion
	    float reff = -shared.gen1[0] / error;
	    error += shared.gen1[0] * reff; // Equivalent to error *= (1 - reff * reff);
    
	    if (threadIdx.x < max_order - 1 - order)
	    {
		float gen1 = shared.gen1[threadIdx.x + 1] + reff * gen0;
		gen0 += shared.gen1[threadIdx.x + 1] * reff;
		shared.gen1[threadIdx.x] = gen1;
	    }

	    // Store prediction error
	    if (threadIdx.x == 0)
		shared.error[order] = error;

	    // Levinson-Durbin recursion
	    shared.ldr[threadIdx.x] += (threadIdx.x < order) * reff * shared.ldr[order - 1 - threadIdx.x] + (threadIdx.x  == order) * reff;

	    // Output coeffs
	    if (threadIdx.x <= order)
		lpcs[shared.lpcOffs + order * 32 + threadIdx.x] = -shared.ldr[order - threadIdx.x];
	}
	// Output prediction error estimates
	if (threadIdx.x < max_order)
	    lpcs[shared.lpcOffs + max_order * 32 + threadIdx.x] = shared.error[threadIdx.x];
    }
}

extern "C" __global__ void cudaComputeLPCLattice(
    FlaCudaSubframeTask *tasks,
    const int taskCount, // tasks per block
    const int *samples,
    const int windowCount,
    const int max_order, // should be <= 12
    float*lpcs
)
{
    __shared__ struct {
	volatile FlaCudaSubframeData task;
	volatile float F[512];
	volatile float arp[32];
	volatile float tmp[256];
	volatile float error[32];
	volatile int lpcOffs;
    } shared;

    // fetch task data
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + taskCount * blockIdx.y))[threadIdx.x];
    if (threadIdx.x == 0)
	shared.lpcOffs = __mul24(__mul24(blockIdx.y + 1, windowCount) - 1, max_order + 1) * 32;
    __syncthreads();

    // F = samples; B = samples
    float s1 = threadIdx.x < shared.task.blocksize ? (samples[shared.task.samplesOffs + threadIdx.x]) / 32768.0f : 0.0f;
    float s2 = threadIdx.x + 256 < shared.task.blocksize ? (samples[shared.task.samplesOffs + threadIdx.x + 256]) / 32768.0f : 0.0f;
    shared.F[threadIdx.x] = s1;
    shared.F[threadIdx.x + 256] = s2;
    __syncthreads();

    shared.tmp[threadIdx.x] = FSQR(s1) + FSQR(s2);
    __syncthreads();
    SUM256(shared.tmp, threadIdx.x, +=);
    __syncthreads();
    float DEN = shared.tmp[0];
    __syncthreads();

    for (int order = 0; order < max_order; order++)
    {
	// reff = F(order+1:frameSize) * B(1:frameSize-order)' / DEN
	int idxF = threadIdx.x + order + 1;
	int idxF2 = idxF + 256;

	shared.tmp[threadIdx.x] = idxF < shared.task.blocksize ? shared.F[idxF] * s1 : 0.0f;
	shared.tmp[threadIdx.x] += idxF2 < shared.task.blocksize ? shared.F[idxF2] * s2 : 0.0f;
	__syncthreads(); 
	SUM256(shared.tmp, threadIdx.x, +=);
	__syncthreads();
	float reff = shared.tmp[0] / DEN;
	__syncthreads();

	// arp(order) = rc(order) = reff
	if (threadIdx.x == 0)
	    shared.arp[order] = reff;
	    //shared.rc[order - 1] = shared.lpc[order - 1][order - 1] = reff;

	// Levinson-Durbin recursion
	// arp(1:order-1) = arp(1:order-1) - reff * arp(order-1:-1:1)
	if (threadIdx.x < order)
	    shared.arp[threadIdx.x] = shared.arp[threadIdx.x] - reff * shared.arp[order - 1 - threadIdx.x];
	
	// Output coeffs
	if (threadIdx.x <= order)
	    lpcs[shared.lpcOffs + order * 32 + threadIdx.x] = shared.arp[order - threadIdx.x];

	// F1 = F(order+1:frameSize) - reff * B(1:frameSize-order)
	// B(1:frameSize-order) = B(1:frameSize-order) - reff * F(order+1:frameSize)
	// F(order+1:frameSize) = F1
	if (idxF < shared.task.blocksize)
	{
	    float f1 = shared.F[idxF];
	    shared.F[idxF] -= reff * s1;
	    s1 -= reff * f1;
	}
	if (idxF2 < shared.task.blocksize)
	{
	    float f2 = shared.F[idxF2];
	    shared.F[idxF2] -= reff * s2;
	    s2 -= reff * f2;
	}

	// DEN = F(order+1:frameSize) * F(order+1:frameSize)' + B(1:frameSize-order) * B(1:frameSize-order)' (BURG)
	shared.tmp[threadIdx.x] = (idxF + 1 < shared.task.blocksize ? FSQR(shared.F[idxF]) + FSQR(s1) : 0);
	shared.tmp[threadIdx.x] += (idxF2 + 1 < shared.task.blocksize ? FSQR(shared.F[idxF2]) + FSQR(s2) : 0);
	__syncthreads();
	SUM256(shared.tmp, threadIdx.x, +=);
	__syncthreads();
	DEN = shared.tmp[0] / 2;
	// shared.PE[order-1] = shared.tmp[0] / 2 / (frameSize - order + 1);
	if (threadIdx.x == 0)
	    shared.error[order] = DEN / (shared.task.blocksize - order);
	__syncthreads();
    }

    // Output prediction error estimates
    if (threadIdx.x < max_order)
	lpcs[shared.lpcOffs + max_order * 32 + threadIdx.x] = shared.error[threadIdx.x];
}

extern "C" __global__ void cudaQuantizeLPC(
    FlaCudaSubframeTask *tasks,
    int taskCount, // tasks per block
    int taskCountLPC, // tasks per set of coeffs (<= 32)
    float*lpcs,
    int max_order, // should be <= 32
    int minprecision,
    int precisions
)
{
    __shared__ struct {
	FlaCudaSubframeData task;
	volatile int tmpi[128];
	volatile int index[64];
	volatile float error[64];
	volatile int lpcOffs;
    } shared;

    if (threadIdx.y == 0)
    {
	// fetch task data
	if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	    ((int*)&shared.task)[threadIdx.x] = ((int*)(tasks + blockIdx.y * taskCount))[threadIdx.x];
	if (threadIdx.x == 0)
	    shared.lpcOffs = (blockIdx.x + blockIdx.y * gridDim.x) * (max_order + 1) * 32;

	shared.index[threadIdx.x] = min(max_order - 1, threadIdx.x);
	shared.error[threadIdx.x] = shared.task.blocksize * 64 + threadIdx.x;
	shared.index[32 + threadIdx.x] = min(max_order - 1, threadIdx.x);
	shared.error[32 + threadIdx.x] = shared.task.blocksize * 64 + threadIdx.x;

        // Select best orders based on Akaike's Criteria

	// Load prediction error estimates
	if (threadIdx.x < max_order)
	    shared.error[threadIdx.x] = shared.task.blocksize * __logf(lpcs[shared.lpcOffs + max_order * 32 + threadIdx.x]) + threadIdx.x * 5.12f * __logf(shared.task.blocksize);
	    //shared.error[threadIdx.x] = shared.task.blocksize * __logf(lpcs[shared.lpcOffs + max_order * 32 + threadIdx.x]) + threadIdx.x * 0.30f * (shared.task.abits + 1) * __logf(shared.task.blocksize);

	// Sort using bitonic sort
	for(int size = 2; size < 64; size <<= 1){
	    //Bitonic merge
	    int ddd = (threadIdx.x & (size / 2)) == 0;
	    for(int stride = size / 2; stride > 0; stride >>= 1){
		//__syncthreads();
		int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
		if ((shared.error[pos] >= shared.error[pos + stride]) == ddd)
		{
		    float t = shared.error[pos];
		    shared.error[pos] = shared.error[pos + stride];
		    shared.error[pos + stride] = t;
		    int t1 = shared.index[pos];
		    shared.index[pos] = shared.index[pos + stride];
		    shared.index[pos + stride] = t1;
		}
	    }
	}

	//ddd == dir for the last bitonic merge step
	{
	    for(int stride = 32; stride > 0; stride >>= 1){
		//__syncthreads();
		int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
		if (shared.error[pos] >= shared.error[pos + stride])
		{
		    float t = shared.error[pos];
		    shared.error[pos] = shared.error[pos + stride];
		    shared.error[pos + stride] = t;
		    int t1 = shared.index[pos];
		    shared.index[pos] = shared.index[pos + stride];
		    shared.index[pos + stride] = t1;
		}
	    }
	}
    }
    __syncthreads();

    const int tid = threadIdx.x + threadIdx.y * 32;

    // Quantization
    for (int i = threadIdx.y; i < taskCountLPC; i += 4)
    {
	int order = shared.index[i >> precisions];
	float lpc = threadIdx.x <= order ? lpcs[shared.lpcOffs + order * 32 + threadIdx.x] : 0.0f;
	// get 15 bits of each coeff
	int coef = __float2int_rn(lpc * (1 << 15));
	// remove sign bits
	shared.tmpi[tid] = coef ^ (coef >> 31);
	// OR reduction
	shared.tmpi[tid] = shared.tmpi[tid] | shared.tmpi[tid + 8] | shared.tmpi[tid + 16] | shared.tmpi[tid + 24];
	shared.tmpi[tid] = shared.tmpi[tid] | shared.tmpi[tid + 2] | shared.tmpi[tid + 4] | shared.tmpi[tid + 6];
	//SUM32(shared.tmpi,tid,|=);
	// choose precision	
	//int cbits = max(3, min(10, 5 + (shared.task.abits >> 1))); //  - __float2int_rn(shared.PE[order - 1])
	int cbits = max(3, min(min(13 - minprecision + (i - ((i >> precisions) << precisions)) - (shared.task.blocksize <= 2304) - (shared.task.blocksize <= 1152) - (shared.task.blocksize <= 576), shared.task.abits), __clz(order) + 1 - shared.task.abits));
	// calculate shift based on precision and number of leading zeroes in coeffs
	int shift = max(0,min(15, __clz(shared.tmpi[threadIdx.y * 32] | shared.tmpi[threadIdx.y * 32 + 1]) - 18 + cbits));
	//if (shared.task.abits + 32 - __clz(order) < shift
	//int shift = max(0,min(15, (shared.task.abits >> 2) - 14 + __clz(shared.tmpi[threadIdx.x & ~31]) + ((32 - __clz(order))>>1)));
	// quantize coeffs with given shift
	coef = max(-(1 << (cbits - 1)), min((1 << (cbits - 1)) -1, __float2int_rn(lpc * (1 << shift))));
	// error correction
	//shared.tmp[threadIdx.x] = (threadIdx.x != 0) * (shared.arp[threadIdx.x - 1]*(1 << shared.task.shift) - shared.task.coefs[threadIdx.x - 1]);
	//shared.task.coefs[threadIdx.x] = max(-(1 << (shared.task.cbits - 1)), min((1 << (shared.task.cbits - 1))-1, __float2int_rn((shared.arp[threadIdx.x]) * (1 << shared.task.shift) + shared.tmp[threadIdx.x])));
	// remove sign bits
	shared.tmpi[tid] = coef ^ (coef >> 31);
	// OR reduction
	shared.tmpi[tid] = shared.tmpi[tid] | shared.tmpi[tid + 8] | shared.tmpi[tid + 16] | shared.tmpi[tid + 24];
	shared.tmpi[tid] = shared.tmpi[tid] | shared.tmpi[tid + 2] | shared.tmpi[tid + 4] | shared.tmpi[tid + 6];
	//SUM32(shared.tmpi,tid,|=);
	// calculate actual number of bits (+1 for sign)
	cbits = 1 + 32 - __clz(shared.tmpi[threadIdx.y * 32] | shared.tmpi[threadIdx.y * 32 + 1]);

	// output shift, cbits and output coeffs
	int taskNo = blockIdx.y * taskCount + blockIdx.x * taskCountLPC + i;
	if (threadIdx.x == 0)
	    tasks[taskNo].data.shift = shift;
	if (threadIdx.x == 0)
	    tasks[taskNo].data.cbits = cbits;
	if (threadIdx.x == 0)
	    tasks[taskNo].data.residualOrder = order + 1;
	if (threadIdx.x <= order)
	    tasks[taskNo].coefs[threadIdx.x] = coef;
    }
}

// blockDim.x == 32
// blockDim.y == 8
extern "C" __global__ void cudaEstimateResidual(
    int*output,
    int*samples,
    FlaCudaSubframeTask *tasks,
    int max_order,
    int partSize // should be blockDim.x * blockDim.y == 256
    )
{
    __shared__ struct {
	int data[32*9];
	volatile int residual[32*8];
	FlaCudaSubframeData task[8];
	int coefs[32*8];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    if (threadIdx.x < sizeof(FlaCudaSubframeData)/sizeof(int))
	((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(&tasks[blockIdx.y * blockDim.y + threadIdx.y]))[threadIdx.x];
    __syncthreads();
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(shared.task[0].blocksize - pos, partSize + max_order);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task[0].samplesOffs + pos + tid] >> shared.task[0].wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task[0].samplesOffs + pos + tid + partSize] >> shared.task[0].wbits : 0;

    __syncthreads();

    shared.residual[tid] = 0;
    shared.coefs[tid] = threadIdx.x < shared.task[threadIdx.y].residualOrder ? tasks[blockIdx.y * blockDim.y + threadIdx.y].coefs[threadIdx.x] : 0;

    const int residualLen = max(0,min(shared.task[0].blocksize - pos - shared.task[threadIdx.y].residualOrder, partSize));
    for (int i = blockDim.y * (shared.task[threadIdx.y].type == Verbatim); i < blockDim.y; i++) // += 32
    {
	// compute residual
	int *co = &shared.coefs[threadIdx.y << 5];
	int ptr = threadIdx.x + (i << 5) + shared.task[threadIdx.y].residualOrder;
	int sum = 0;
	for (int c = -shared.task[threadIdx.y].residualOrder; c < 0; c++)
	    sum += __mul24(shared.data[ptr + c], *(co++));
	sum = shared.data[ptr] - (sum >> shared.task[threadIdx.y].shift);
	shared.residual[tid] += __mul24(ptr < dataLen, min(0x7fffff,(sum << 1) ^ (sum >> 31)));
    }

    shared.residual[tid] = shared.residual[tid] + shared.residual[tid + 8] + shared.residual[tid + 16] + shared.residual[tid + 24];
    shared.residual[tid] = shared.residual[tid] + shared.residual[tid + 2] + shared.residual[tid + 4] + shared.residual[tid + 6];
    if (threadIdx.x == 0)
	output[(blockIdx.y * blockDim.y + threadIdx.y) * 64 + blockIdx.x] = shared.residual[tid] + shared.residual[tid + 1];
}

extern "C" __global__ void cudaEstimateResidual1(
    int*output,
    int*samples,
    FlaCudaSubframeTask *tasks,
    int max_order,
    int partSize // should be blockDim.x * blockDim.y == 256
    )
{
    __shared__ struct {
	int data[32*9];
	volatile int residual[32*8];
	FlaCudaSubframeTask task;
	volatile int pos;
	volatile int dataLen;
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    if (tid < sizeof(shared.task)/sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    if (tid == 0)
    {
	shared.pos = blockIdx.x * partSize; 
	shared.dataLen =  min(shared.task.data.blocksize - shared.pos, partSize + shared.task.data.residualOrder);
    }
    __syncthreads();

    // fetch samples
    shared.data[tid] = tid < shared.dataLen ? samples[shared.task.data.samplesOffs + shared.pos + tid] >> shared.task.data.wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < shared.dataLen ? samples[shared.task.data.samplesOffs + shared.pos + tid + partSize] >> shared.task.data.wbits : 0;
    __syncthreads();

    // compute residual
    int *co = &shared.task.coefs[0];
    int ptr = tid + shared.task.data.residualOrder;
    int sum = 0;
    for (int c = -shared.task.data.residualOrder; c < 0; c++)
	sum += __mul24(shared.data[ptr + c], *(co++));
    sum = shared.data[ptr] - (sum >> shared.task.data.shift);
    shared.residual[tid] = __mul24(ptr < shared.dataLen, min(0x7fffff,(sum << 1) ^ (sum >> 31)));
    __syncthreads();
    SUM256(shared.residual, tid, +=);
    if (tid == 0)
	output[blockIdx.y * 64 + blockIdx.x] = shared.residual[0];
}

extern "C" __global__ void cudaEstimateResidual8(
    int*output,
    int*samples,
    FlaCudaSubframeTask *tasks,
    int max_order,
    int partSize // should be blockDim.x * blockDim.y == 256
    )
{
    __shared__ struct {
	volatile int data[32*9];
	volatile int residual[32*8];
	FlaCudaSubframeData task[8];
	int coefs[32*8];
	volatile int pos;
	volatile int dataLen;
	volatile int dataOffs;
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    const int taskNo = FASTMUL(blockIdx.y, blockDim.y) + threadIdx.y;
    if (threadIdx.x < sizeof(FlaCudaSubframeData)/sizeof(int))
	((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(&tasks[taskNo]))[threadIdx.x];
    const int ro = shared.task[threadIdx.y].residualOrder;
    shared.coefs[tid] = threadIdx.x < ro ? tasks[taskNo].coefs[threadIdx.x] : 0;
    if (tid == 0)
    {
	shared.pos = FASTMUL(blockIdx.x, partSize);
	shared.dataLen =  min(shared.task[0].blocksize - shared.pos, partSize + max_order);
	shared.dataOffs = shared.task[0].samplesOffs + shared.pos;
    }
    __syncthreads();

    // fetch samples
    if (tid < shared.dataLen)
	shared.data[tid] = samples[shared.dataOffs + tid] >> shared.task[0].wbits;
    if (tid + partSize < shared.dataLen)
	shared.data[tid + partSize] = samples[shared.dataOffs + tid + partSize] >> shared.task[0].wbits;

    __syncthreads();

    const int residualLen = max(0,min(shared.dataLen - ro, partSize));
    const int ptr2 = threadIdx.y << 5;
    int s = 0;
    for (int ptr = threadIdx.x; ptr < residualLen; ptr += 32)
    {
	// compute residual
	int sum = 
	    __mul24(shared.data[ptr + 0], shared.coefs[ptr2 + 0]) +
	    __mul24(shared.data[ptr + 1], shared.coefs[ptr2 + 1]) +
	    __mul24(shared.data[ptr + 2], shared.coefs[ptr2 + 2]) +
	    __mul24(shared.data[ptr + 3], shared.coefs[ptr2 + 3]);
	sum +=
	    __mul24(shared.data[ptr + 4], shared.coefs[ptr2 + 4]) +
	    __mul24(shared.data[ptr + 5], shared.coefs[ptr2 + 5]) +
	    __mul24(shared.data[ptr + 6], shared.coefs[ptr2 + 6]) +
	    __mul24(shared.data[ptr + 7], shared.coefs[ptr2 + 7]);
	sum = shared.data[ptr + ro] - (sum >> shared.task[threadIdx.y].shift);
	s += min(0x7fffff,(sum << 1) ^ (sum >> 31));
    }

    shared.residual[tid] = s;
    shared.residual[tid] = shared.residual[tid] + shared.residual[tid + 8] + shared.residual[tid + 16] + shared.residual[tid + 24];
    shared.residual[tid] = shared.residual[tid] + shared.residual[tid + 2] + shared.residual[tid + 4] + shared.residual[tid + 6];
    if (threadIdx.x == 0)
	output[(blockIdx.y * blockDim.y + threadIdx.y) * 64 + blockIdx.x] = shared.residual[tid] + shared.residual[tid + 1];
}

extern "C" __global__ void cudaEstimateResidual12(
    int*output,
    int*samples,
    FlaCudaSubframeTask *tasks,
    int max_order,
    int partSize // should be blockDim.x * blockDim.y == 256
    )
{
    __shared__ struct {
	volatile int data[32*9];
	volatile int residual[32*8];
	FlaCudaSubframeData task[8];
	int coefs[8*32];
	volatile int pos;
	volatile int dataLen;
	volatile int dataOffs;
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    const int taskNo = FASTMUL(blockIdx.y, blockDim.y) + threadIdx.y;
    if (threadIdx.x < sizeof(FlaCudaSubframeData)/sizeof(int))
	((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(&tasks[taskNo]))[threadIdx.x];
    const int ro = shared.task[threadIdx.y].residualOrder;
    shared.coefs[tid] = threadIdx.x < ro ? tasks[taskNo].coefs[threadIdx.x] : 0;
    if (tid == 0)
    {
	shared.pos = FASTMUL(blockIdx.x, partSize);
	shared.dataLen =  min(shared.task[0].blocksize - shared.pos, partSize + max_order);
	shared.dataOffs = shared.task[0].samplesOffs + shared.pos;
    }
    __syncthreads();

    // fetch samples
    if (tid < shared.dataLen)
	shared.data[tid] = samples[shared.dataOffs + tid] >> shared.task[0].wbits;
    if (tid + partSize < shared.dataLen)
	shared.data[tid + partSize] = samples[shared.dataOffs + tid + partSize] >> shared.task[0].wbits;

    __syncthreads();

    int residualLen = max(0,min(shared.dataLen - ro, partSize));
    const int ptr2 = threadIdx.y << 5;
    int s = 0;
    for (int ptr = threadIdx.x; ptr < residualLen; ptr += 32)
    {
	// compute residual
	int sum =
    	    FASTMUL(shared.data[ptr + 0], shared.coefs[ptr2 + 0]) +
	    FASTMUL(shared.data[ptr + 1], shared.coefs[ptr2 + 1]) +
	    FASTMUL(shared.data[ptr + 2], shared.coefs[ptr2 + 2]) +
	    FASTMUL(shared.data[ptr + 3], shared.coefs[ptr2 + 3]);
	sum += 
	    FASTMUL(shared.data[ptr + 4], shared.coefs[ptr2 + 4]) +
	    FASTMUL(shared.data[ptr + 5], shared.coefs[ptr2 + 5]) +
	    FASTMUL(shared.data[ptr + 6], shared.coefs[ptr2 + 6]) +
	    FASTMUL(shared.data[ptr + 7], shared.coefs[ptr2 + 7]);
	sum +=
	    FASTMUL(shared.data[ptr + 8], shared.coefs[ptr2 + 8]) +
	    FASTMUL(shared.data[ptr + 9], shared.coefs[ptr2 + 9]) +
	    FASTMUL(shared.data[ptr + 10], shared.coefs[ptr2 + 10]) +
	    FASTMUL(shared.data[ptr + 11], shared.coefs[ptr2 + 11]);
	sum = shared.data[ptr + ro] - (sum >> shared.task[threadIdx.y].shift);
	s += min(0x7fffff,(sum << 1) ^ (sum >> 31));
    }

    shared.residual[tid] = s;
    shared.residual[tid] = shared.residual[tid] + shared.residual[tid + 8] + shared.residual[tid + 16] + shared.residual[tid + 24];
    shared.residual[tid] = shared.residual[tid] + shared.residual[tid + 2] + shared.residual[tid + 4] + shared.residual[tid + 6];
    if (threadIdx.x == 0)
	output[(blockIdx.y * blockDim.y + threadIdx.y) * 64 + blockIdx.x] = shared.residual[tid] + shared.residual[tid + 1];
}

extern "C" __global__ void cudaChooseBestMethod(
    FlaCudaSubframeTask *tasks,
    int *residual,
    int partSize,
    int partCount, // <= blockDim.y (256)
    int taskCount
    )
{
    __shared__ struct {
	volatile int index[128];
	volatile int length[256];
	volatile int partLen[256];
	volatile FlaCudaSubframeTask task[8];
    } shared;
    const int tid = threadIdx.x + threadIdx.y * 32;
    
    shared.length[tid] = 0x7fffffff;
    for (int task = 0; task < taskCount; task += blockDim.y)
	if (task + threadIdx.y < taskCount)
	{
	    // fetch task data
	    ((int*)&shared.task[threadIdx.y])[threadIdx.x] = ((int*)(tasks + task + threadIdx.y + taskCount * blockIdx.y))[threadIdx.x];

	    int sum = 0;
	    for (int pos = threadIdx.x; pos < partCount; pos += blockDim.x)
	    {
		// fetch part sum
		int psum = residual[pos + 64 * (task + threadIdx.y + taskCount * blockIdx.y)];
		// calculate part size
		int residualLen = max(0,min(shared.task[threadIdx.y].data.blocksize - FASTMUL(pos, partSize) - shared.task[threadIdx.y].data.residualOrder, partSize));
		residualLen = FASTMUL(residualLen, shared.task[threadIdx.y].data.type != Constant || psum != 0);
		// calculate rice parameter
		int k = max(0, min(14, __float2int_rz(__log2f((psum + 0.000001f) / (residualLen + 0.000001f) + 0.5f))));
		// calculate part bit length
		sum += FASTMUL(residualLen, k + 1) + (psum >> k);
	    }
	    shared.partLen[tid] = sum;

	    // length sum: reduction in shared mem
	    shared.partLen[tid] += shared.partLen[tid + 16];
	    shared.partLen[tid] += shared.partLen[tid + 8];
	    shared.partLen[tid] += shared.partLen[tid + 4];
	    shared.partLen[tid] += shared.partLen[tid + 2];
	    shared.partLen[tid] += shared.partLen[tid + 1];
	    // return sum
	    if (threadIdx.x == 0)
	    {
		int obits = shared.task[threadIdx.y].data.obits - shared.task[threadIdx.y].data.wbits;
		shared.length[task + threadIdx.y] =
		    min(obits * shared.task[threadIdx.y].data.blocksize,
			shared.task[threadIdx.y].data.type == Fixed ? shared.task[threadIdx.y].data.residualOrder * obits + 6 + (4 * partCount/2) + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].data.type == LPC ? shared.task[threadIdx.y].data.residualOrder * obits + 4 + 5 + shared.task[threadIdx.y].data.residualOrder * shared.task[threadIdx.y].data.cbits + 6 + (4 * partCount/2)/* << porder */ + shared.partLen[threadIdx.y * 32] :
			shared.task[threadIdx.y].data.type == Constant ? obits * (1 + shared.task[threadIdx.y].data.blocksize * (shared.partLen[threadIdx.y * 32] != 0)) : 
			obits * shared.task[threadIdx.y].data.blocksize);
	    }
	}
    //shared.index[threadIdx.x] = threadIdx.x;
    //shared.length[threadIdx.x] = (threadIdx.x < taskCount) ? tasks[threadIdx.x + taskCount * blockIdx.y].size : 0x7fffffff;

    __syncthreads();

    if (tid < taskCount)
	tasks[tid + taskCount * blockIdx.y].data.size = shared.length[tid];

    __syncthreads();
    int l1 = shared.length[tid];
    if (tid < 128)
    {
	int l2 = shared.length[tid + 128];
	shared.index[tid] = tid + ((l2 < l1) << 7);
	shared.length[tid] = l1 = min(l1, l2);
    }
    __syncthreads();
    if (tid < 64)
    {
	int l2 = shared.length[tid + 64];
	shared.index[tid] = shared.index[tid + ((l2 < l1) << 6)];
	shared.length[tid] = l1 = min(l1, l2);
    }
    __syncthreads();
    if (tid < 32)
    {
#pragma unroll 5
	for (int sh = 5; sh > 0; sh --)
	{
	    int l2 = shared.length[tid + (1 << sh)];
	    shared.index[tid] = shared.index[tid + ((l2 < l1) << sh)];
	    shared.length[tid] = l1 = min(l1, l2);
	}
	if (tid == 0)
	    tasks[taskCount * blockIdx.y].data.best_index = taskCount * blockIdx.y + shared.index[shared.length[1] < shared.length[0]];
    }
}

extern "C" __global__ void cudaCopyBestMethod(
    FlaCudaSubframeTask *tasks_out,
    FlaCudaSubframeTask *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index;
    } shared;
    if (threadIdx.x == 0)
	shared.best_index = tasks[count * blockIdx.y].data.best_index;
    __syncthreads();
    if (threadIdx.x < sizeof(FlaCudaSubframeTask)/sizeof(int))
	((int*)(tasks_out + blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.best_index))[threadIdx.x];
}

extern "C" __global__ void cudaCopyBestMethodStereo(
    FlaCudaSubframeTask *tasks_out,
    FlaCudaSubframeTask *tasks,
    int count
    )
{
    __shared__ struct {
	int best_index[4];
	int best_size[4];
	int lr_index[2];
    } shared;
    if (threadIdx.x < 4)
	shared.best_index[threadIdx.x] = tasks[count * (blockIdx.y * 4 + threadIdx.x)].data.best_index;
    if (threadIdx.x < 4)
	shared.best_size[threadIdx.x] = tasks[shared.best_index[threadIdx.x]].data.size;
    __syncthreads();
    if (threadIdx.x == 0)
    {
	int bitsBest = 0x7fffffff;
	if (bitsBest > shared.best_size[2] + shared.best_size[3]) // MidSide
	{
	    bitsBest = shared.best_size[2] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[2];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[3] + shared.best_size[1]) // RightSide
	{
	    bitsBest = shared.best_size[3] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[3];
	    shared.lr_index[1] = shared.best_index[1];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[3]) // LeftSide
	{
	    bitsBest = shared.best_size[0] + shared.best_size[3];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[3];
	}
	if (bitsBest > shared.best_size[0] + shared.best_size[1]) // LeftRight
	{
	    bitsBest = shared.best_size[0] + shared.best_size[1];
	    shared.lr_index[0] = shared.best_index[0];
	    shared.lr_index[1] = shared.best_index[1];
	}
    }
    __syncthreads();
    if (threadIdx.x < sizeof(FlaCudaSubframeTask)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y))[threadIdx.x] = ((int*)(tasks + shared.lr_index[0]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y].data.residualOffs = tasks[shared.best_index[0]].data.residualOffs;
    if (threadIdx.x < sizeof(FlaCudaSubframeTask)/sizeof(int))
	((int*)(tasks_out + 2 * blockIdx.y + 1))[threadIdx.x] = ((int*)(tasks + shared.lr_index[1]))[threadIdx.x];
    if (threadIdx.x == 0)
	tasks_out[2 * blockIdx.y + 1].data.residualOffs = tasks[shared.best_index[1]].data.residualOffs;
}

extern "C" __global__ void cudaEncodeResidual(
    int*output,
    int*samples,
    FlaCudaSubframeTask *tasks
    )
{
    __shared__ struct {
	int data[256 + 32];
	FlaCudaSubframeTask task;
    } shared;
    const int tid = threadIdx.x;
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.y]))[threadIdx.x];
    __syncthreads();
    const int partSize = blockDim.x;
    const int pos = blockIdx.x * partSize;
    const int dataLen = min(shared.task.data.blocksize - pos, partSize + shared.task.data.residualOrder);

    // fetch samples
    shared.data[tid] = tid < dataLen ? samples[shared.task.data.samplesOffs + pos + tid] >> shared.task.data.wbits : 0;
    if (tid < 32) shared.data[tid + partSize] = tid + partSize < dataLen ? samples[shared.task.data.samplesOffs + pos + tid + partSize] >> shared.task.data.wbits : 0;
    const int residualLen = max(0,min(shared.task.data.blocksize - pos - shared.task.data.residualOrder, partSize));

    __syncthreads();    
    // compute residual
    int sum = 0;
    for (int c = 0; c < shared.task.data.residualOrder; c++)
	sum += __mul24(shared.data[tid + c], shared.task.coefs[c]);
    __syncthreads();
    shared.data[tid + shared.task.data.residualOrder] -= (sum >> shared.task.data.shift);
    __syncthreads();
    if (tid >= shared.task.data.residualOrder && tid < residualLen + shared.task.data.residualOrder)
	output[shared.task.data.residualOffs + pos + tid] = shared.data[tid];
    if (tid + 256 < residualLen + shared.task.data.residualOrder)
	output[shared.task.data.residualOffs + pos + tid + 256] = shared.data[tid + 256];
}

extern "C" __global__ void cudaCalcPartition(
    int* partition_lengths,
    int* residual,
    int* samples,
    FlaCudaSubframeTask *tasks,
    int max_porder, // <= 8
    int psize, // == (shared.task.data.blocksize >> max_porder), < 256
    int parts_per_block // == 256 / psize, > 0, <= 16
    )
{
    __shared__ struct {
	int data[256+32];
	FlaCudaSubframeTask task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    const int parts = min(parts_per_block, (1 << max_porder) - blockIdx.x * parts_per_block);
    const int offs = blockIdx.x * psize * parts_per_block + tid;

    // fetch samples
    if (tid < 32) shared.data[tid] = min(offs, tid + shared.task.data.residualOrder) >= 32 ? samples[shared.task.data.samplesOffs + offs - 32] >> shared.task.data.wbits : 0;
    shared.data[32 + tid] = tid < parts * psize ? samples[shared.task.data.samplesOffs + offs] >> shared.task.data.wbits : 0;
    __syncthreads();

    // compute residual
    int s = 0;
    for (int c = -shared.task.data.residualOrder; c < 0; c++)
	s += __mul24(shared.data[32 + tid + c], shared.task.coefs[shared.task.data.residualOrder + c]);
    s = shared.data[32 + tid] - (s >> shared.task.data.shift);

    if (offs >= shared.task.data.residualOrder && tid < parts * psize)
	residual[shared.task.data.residualOffs + offs] = s;
    else
	s = 0;

    // convert to unsigned
    s = min(0xfffff, (s << 1) ^ (s >> 31));

    //__syncthreads();
    //shared.data[tid] = s;
    //__syncthreads();

    //shared.data[tid] = (shared.data[tid] & (0x0000ffff << (tid & 16))) | (((shared.data[tid ^ 16] & (0x0000ffff << (tid & 16))) << (~tid & 16)) >> (tid & 16));
    //shared.data[tid] = (shared.data[tid] & (0x00ff00ff << (tid & 8))) | (((shared.data[tid ^ 8] & (0x00ff00ff << (tid & 8))) << (~tid & 8)) >> (tid & 8));
    //shared.data[tid] = (shared.data[tid] & (0x0f0f0f0f << (tid & 4))) | (((shared.data[tid ^ 4] & (0x0f0f0f0f << (tid & 4))) << (~tid & 4)) >> (tid & 4));
    //shared.data[tid] = (shared.data[tid] & (0x33333333 << (tid & 2))) | (((shared.data[tid ^ 2] & (0x33333333 << (tid & 2))) << (~tid & 2)) >> (tid & 2));
    //shared.data[tid] = (shared.data[tid] & (0x55555555 << (tid & 1))) | (((shared.data[tid ^ 1] & (0x55555555 << (tid & 1))) << (~tid & 1)) >> (tid & 1));
    //shared.data[tid] = __popc(shared.data[tid]);

    __syncthreads();
    shared.data[tid + (tid / psize)] = s;
    //shared.data[tid] = s;
    __syncthreads();

    s = (psize - shared.task.data.residualOrder * (threadIdx.x + blockIdx.x == 0)) * (threadIdx.y + 1);
    int dpos = __mul24(threadIdx.x, psize + 1);
    //int dpos = __mul24(threadIdx.x, psize);
    // calc number of unary bits for part threadIdx.x with rice parameter threadIdx.y
#pragma unroll 0
    for (int i = 0; i < psize; i++)
	s += shared.data[dpos + i] >> threadIdx.y;

    // output length
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));
    if (threadIdx.y <= 14 && threadIdx.x < parts)
	partition_lengths[pos + blockIdx.x * parts_per_block + threadIdx.x] = s;
}

extern "C" __global__ void cudaCalcPartition16(
    int* partition_lengths,
    int* residual,
    int* samples,
    FlaCudaSubframeTask *tasks,
    int max_porder, // <= 8
    int psize, // == 16
    int parts_per_block // == 16
    )
{
    __shared__ struct {
	int data[256+32];
	FlaCudaSubframeTask task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    const int offs = (blockIdx.x << 8) + tid;

    // fetch samples
    if (tid < 32) shared.data[tid] = min(offs, tid + shared.task.data.residualOrder) >= 32 ? samples[shared.task.data.samplesOffs + offs - 32] >> shared.task.data.wbits : 0;
    shared.data[32 + tid] = samples[shared.task.data.samplesOffs + offs] >> shared.task.data.wbits;
 //   if (tid < 32 && tid >= shared.task.data.residualOrder)
	//shared.task.coefs[tid] = 0;
    __syncthreads();

    // compute residual
    int s = 0;
    for (int c = -shared.task.data.residualOrder; c < 0; c++)
	s += __mul24(shared.data[32 + tid + c], shared.task.coefs[shared.task.data.residualOrder + c]);
 //   int spos = 32 + tid - shared.task.data.residualOrder;
 //   int s=
	//__mul24(shared.data[spos + 0], shared.task.coefs[0]) + __mul24(shared.data[spos + 1], shared.task.coefs[1]) + 
	//__mul24(shared.data[spos + 2], shared.task.coefs[2]) + __mul24(shared.data[spos + 3], shared.task.coefs[3]) + 
	//__mul24(shared.data[spos + 4], shared.task.coefs[4]) + __mul24(shared.data[spos + 5], shared.task.coefs[5]) + 
	//__mul24(shared.data[spos + 6], shared.task.coefs[6]) + __mul24(shared.data[spos + 7], shared.task.coefs[7]) +
	//__mul24(shared.data[spos + 8], shared.task.coefs[8]) + __mul24(shared.data[spos + 9], shared.task.coefs[9]) + 
	//__mul24(shared.data[spos + 10], shared.task.coefs[10]) + __mul24(shared.data[spos + 11], shared.task.coefs[11]) +
	//__mul24(shared.data[spos + 12], shared.task.coefs[12]) + __mul24(shared.data[spos + 13], shared.task.coefs[13]) + 
	//__mul24(shared.data[spos + 14], shared.task.coefs[14]) + __mul24(shared.data[spos + 15], shared.task.coefs[15]);
    s = shared.data[32 + tid] - (s >> shared.task.data.shift);

    if (blockIdx.x != 0 || tid >= shared.task.data.residualOrder)
	residual[shared.task.data.residualOffs + (blockIdx.x << 8) + tid] = s;
    else
	s = 0;

    // convert to unsigned
    s = min(0xfffff, (s << 1) ^ (s >> 31));
    __syncthreads();
    shared.data[tid + threadIdx.y] = s;
    __syncthreads();

    // calc number of unary bits for part threadIdx.x with rice parameter threadIdx.y
    int dpos = __mul24(threadIdx.x, 17);
    int sum =
	(shared.data[dpos + 0] >> threadIdx.y) + (shared.data[dpos + 1] >> threadIdx.y) + 
	(shared.data[dpos + 2] >> threadIdx.y) + (shared.data[dpos + 3] >> threadIdx.y) + 
	(shared.data[dpos + 4] >> threadIdx.y) + (shared.data[dpos + 5] >> threadIdx.y) + 
	(shared.data[dpos + 6] >> threadIdx.y) + (shared.data[dpos + 7] >> threadIdx.y) + 
	(shared.data[dpos + 8] >> threadIdx.y) + (shared.data[dpos + 9] >> threadIdx.y) + 
	(shared.data[dpos + 10] >> threadIdx.y) + (shared.data[dpos + 11] >> threadIdx.y) + 
	(shared.data[dpos + 12] >> threadIdx.y) + (shared.data[dpos + 13] >> threadIdx.y) + 
	(shared.data[dpos + 14] >> threadIdx.y) + (shared.data[dpos + 15] >> threadIdx.y);

    // output length
    const int pos = ((15 * blockIdx.y + threadIdx.y) << (max_porder + 1)) + (blockIdx.x << 4) + threadIdx.x;
    if (threadIdx.y <= 14)
	partition_lengths[pos] = sum + (16 - shared.task.data.residualOrder * (threadIdx.x + blockIdx.x == 0)) * (threadIdx.y + 1);
}

extern "C" __global__ void cudaCalcLargePartition(
    int* partition_lengths,
    int* residual,
    int* samples,
    FlaCudaSubframeTask *tasks,
    int max_porder, // <= 8
    int psize, // == >= 128
    int parts_per_block // == 1
    )
{
    __shared__ struct {
	int data[256];
	volatile int length[256];
	FlaCudaSubframeTask task;
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 4);
    if (tid < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[tid] = ((int*)(&tasks[blockIdx.y]))[tid];
    __syncthreads();

    int sum = 0;
    for (int pos = 0; pos < psize; pos += 256)
    {
	// fetch residual
	int offs = blockIdx.x * psize + pos + tid;
	int s = (offs >= shared.task.data.residualOrder && pos + tid < psize) ? residual[shared.task.data.residualOffs + offs] : 0;
	// convert to unsigned
	shared.data[tid] = min(0xfffff, (s << 1) ^ (s >> 31));
	__syncthreads();

	// calc number of unary bits for each residual sample with each rice parameter
#pragma unroll 0
	for (int i = threadIdx.x; i < min(psize,256); i += 16)
	    // for sample (i + threadIdx.x) with this rice parameter (threadIdx.y)
	    sum += shared.data[i] >> threadIdx.y;
	__syncthreads();
    }
    shared.length[tid] = min(0xfffff,sum);
    SUM16(shared.length,tid,+=);

    // output length
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));
    if (threadIdx.y <= 14 && threadIdx.x == 0)
	partition_lengths[pos + blockIdx.x] = min(0xfffff,shared.length[tid]) + (psize - shared.task.data.residualOrder * (blockIdx.x == 0)) * (threadIdx.y + 1);
}

// Sums partition lengths for a certain k == blockIdx.x
// Requires 128 threads
extern "C" __global__ void cudaSumPartition(
    int* partition_lengths,
    int max_porder
    )
{
    __shared__ struct {
	volatile int data[512+32]; // max_porder <= 8, data length <= 1 << 9.
    } shared;

    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (blockIdx.x << (max_porder + 1));

    // fetch partition lengths
    shared.data[threadIdx.x] = threadIdx.x < (1 << max_porder) ? partition_lengths[pos + threadIdx.x] : 0;
    shared.data[blockDim.x + threadIdx.x] = blockDim.x + threadIdx.x < (1 << max_porder) ? partition_lengths[pos + blockDim.x + threadIdx.x] : 0;
    __syncthreads();

    int in_pos = (threadIdx.x << 1);
    int out_pos = (1 << max_porder) + threadIdx.x;
    int bs;
    for (bs = 1 << (max_porder - 1); bs > 32; bs >>= 1)
    {
	if (threadIdx.x < bs) shared.data[out_pos] = shared.data[in_pos] + shared.data[in_pos + 1];
	in_pos += bs << 1;
	out_pos += bs;
	__syncthreads();
    }
    if (threadIdx.x < 32)
    for (; bs > 0; bs >>= 1)
    {
	shared.data[out_pos] = shared.data[in_pos] + shared.data[in_pos + 1];
	in_pos += bs << 1;
	out_pos += bs;
    }
    __syncthreads();
    if (threadIdx.x < (1 << max_porder))
	partition_lengths[pos + (1 << max_porder) + threadIdx.x] = shared.data[(1 << max_porder) + threadIdx.x];
    if (blockDim.x + threadIdx.x < (1 << max_porder))
	partition_lengths[pos + (1 << max_porder) + blockDim.x + threadIdx.x] = shared.data[(1 << max_porder) + blockDim.x + threadIdx.x];
}

// Finds optimal rice parameter for up to 16 partitions at a time.
// Requires 16x16 threads
extern "C" __global__ void cudaFindRiceParameter(
    int* rice_parameters,
    int* partition_lengths,
    int max_porder
    )
{
    __shared__ struct {
	volatile int length[256];
	volatile int index[256];
    } shared;
    const int tid = threadIdx.x + (threadIdx.y << 5);
    const int parts = min(32, 2 << max_porder);
    const int pos = (15 << (max_porder + 1)) * blockIdx.y + (threadIdx.y << (max_porder + 1));

    // read length for 32 partitions
    int l1 = (threadIdx.x < parts) ? partition_lengths[pos + blockIdx.x * 32 + threadIdx.x] : 0xffffff;
    int l2 = (threadIdx.y + 8 <= 14 && threadIdx.x < parts) ? partition_lengths[pos + (8 << (max_porder + 1)) + blockIdx.x * 32 + threadIdx.x] : 0xffffff;
    // find best rice parameter
    shared.index[tid] = threadIdx.y + ((l2 < l1) << 3);
    shared.length[tid] = l1 = min(l1, l2);
    __syncthreads();
#pragma unroll 3
    for (int sh = 7; sh >= 5; sh --)
    {
	if (tid < (1 << sh))
	{
	    l2 = shared.length[tid + (1 << sh)];
	    shared.index[tid] = shared.index[tid + ((l2 < l1) << sh)];
	    shared.length[tid] = l1 = min(l1, l2);
	}    
	__syncthreads();
    }
    if (tid < parts)
    {
	// output rice parameter
	rice_parameters[(blockIdx.y << (max_porder + 2)) + blockIdx.x * parts + tid] = shared.index[tid];
	// output length
	rice_parameters[(blockIdx.y << (max_porder + 2)) + (1 << (max_porder + 1)) + blockIdx.x * parts + tid] = shared.length[tid];
    }
}

extern "C" __global__ void cudaFindPartitionOrder(
    int* best_rice_parameters,
    FlaCudaSubframeTask *tasks,
    int* rice_parameters,
    int max_porder
    )
{
    __shared__ struct {
	int data[512];
	volatile int tmp[256];
	int length[32];
	int index[32];
	//char4 ch[64];
	FlaCudaSubframeTask task;
    } shared;
    const int pos = (blockIdx.y << (max_porder + 2)) + (2 << max_porder);
    if (threadIdx.x < sizeof(shared.task) / sizeof(int))
	((int*)&shared.task)[threadIdx.x] = ((int*)(&tasks[blockIdx.y]))[threadIdx.x];
    // fetch partition lengths
    shared.data[threadIdx.x] = threadIdx.x < (2 << max_porder) ? rice_parameters[pos + threadIdx.x] : 0;
    shared.data[threadIdx.x + 256] = threadIdx.x + 256 < (2 << max_porder) ? rice_parameters[pos + 256 + threadIdx.x] : 0;
    __syncthreads();

    for (int porder = max_porder; porder >= 0; porder--)
    {
	shared.tmp[threadIdx.x] = (threadIdx.x < (1 << porder)) * shared.data[(2 << max_porder) - (2 << porder) + threadIdx.x];
	__syncthreads();
	SUM256(shared.tmp, threadIdx.x, +=);
	if (threadIdx.x == 0)
	    shared.length[porder] = shared.tmp[0] + (4 << porder);
	__syncthreads();
    }

    if (threadIdx.x < 32)
    {
	shared.index[threadIdx.x] = threadIdx.x;
	if (threadIdx.x > max_porder)
	    shared.length[threadIdx.x] = 0xfffffff;
	int l1 = shared.length[threadIdx.x];
    #pragma unroll 4
	for (int sh = 3; sh >= 0; sh --)
	{
	    int l2 = shared.length[threadIdx.x + (1 << sh)];
	    shared.index[threadIdx.x] = shared.index[threadIdx.x + ((l2 < l1) << sh)];
	    shared.length[threadIdx.x] = l1 = min(l1, l2);
	}
	if (threadIdx.x == 0)
	    tasks[blockIdx.y].data.porder = shared.index[0];
	if (threadIdx.x == 0)
	{
	    int obits = shared.task.data.obits - shared.task.data.wbits;	    
	    tasks[blockIdx.y].data.size =
		shared.task.data.type == Fixed ? shared.task.data.residualOrder * obits + 6 + l1 :
		shared.task.data.type == LPC ? shared.task.data.residualOrder * obits + 6 + l1 + 4 + 5 + shared.task.data.residualOrder * shared.task.data.cbits :
		shared.task.data.type == Constant ? obits : obits * shared.task.data.blocksize;
	}
    }
    __syncthreads();
    int porder = shared.index[0];
    if (threadIdx.x < (1 << porder))
	best_rice_parameters[(blockIdx.y << max_porder) + threadIdx.x] = rice_parameters[pos - (2 << porder) + threadIdx.x];
    // FIXME: should be bytes?
 //   if (threadIdx.x < (1 << porder))
	//shared.tmp[threadIdx.x] = rice_parameters[pos - (2 << porder) + threadIdx.x];
 //   __syncthreads();
 //   if (threadIdx.x < max(1, (1 << porder) >> 2))
 //   {
	//char4 ch;
	//ch.x = shared.tmp[(threadIdx.x << 2)];
	//ch.y = shared.tmp[(threadIdx.x << 2) + 1];
	//ch.z = shared.tmp[(threadIdx.x << 2) + 2];
	//ch.w = shared.tmp[(threadIdx.x << 2) + 3];
	//shared.ch[threadIdx.x] = ch
 //   }	
 //   __syncthreads();
 //   if (threadIdx.x < max(1, (1 << porder) >> 2))
	//best_rice_parameters[(blockIdx.y << max_porder) + threadIdx.x] = shared.ch[threadIdx.x];
}

#endif

#if 0
    if (threadIdx.x < order)
    {
	for (int i = 0; i < order; i++)
	    if (threadIdx.x >= i)
		sum[threadIdx.x - i] += coefs[threadIdx.x] * sample[order - i - 1];
	for (int i = order; i < blocksize; i++)
	{
	    if (!threadIdx.x) sample[order + i] = s = residual[order + i] + (sum[order + i] >> shift);
	    sum[threadIdx.x + i + 1] += coefs[threadIdx.x] * s;
	}
    }
#endif